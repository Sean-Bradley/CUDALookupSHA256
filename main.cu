#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <ctime>

using namespace std;

extern "C" __global__ static void kernel(unsigned char* output, unsigned char* query, int queryLength, int N);
__device__ unsigned char* SHA256Compute(unsigned char* output, unsigned char* query, int queryLength, int idx, int idy, int idz);
__device__ bool SHATransform(unsigned int* expandedBuffer, int expandedBufferLen0, unsigned int* state, unsigned char* block, int blockLen0, unsigned int* _K, int _KLen0);
__device__ unsigned int RotateRight(unsigned int x, int n);
__device__ unsigned int Ch(unsigned int x, unsigned int y, unsigned int z);
__device__ unsigned int Maj(unsigned int x, unsigned int y, unsigned int z);
__device__ unsigned int sigma_0(unsigned int x);
__device__ unsigned int sigma_1(unsigned int x);
__device__ unsigned int Sigma_0(unsigned int x);
__device__ unsigned int Sigma_1(unsigned int x);
__device__ void DWORDToBigEndian(unsigned char* block, unsigned int* x, int digits);
__device__ void DWORDFromBigEndian(unsigned int* x, int xLen0, int digits, unsigned char* block, int blockLen0);
__device__ void CopyArray(unsigned char* SourceArray, int SourceArrayLen0, int SourceIndex, unsigned char* DestinationArray, int DestinationArrayLen0, int DestinationIndex, int Length);
extern "C" __global__ static void kernel(unsigned char* output, unsigned char* query, int queryLength, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		/*for(int v1 = 0x73; v1 < 0x74; v1++)
		{*/
		SHA256Compute(output, query, queryLength, idx & 0x000000FF, idx >> 8, 0x73);
		//}
	}
}
//__device__
__device__ unsigned char* SHA256Compute(unsigned char* output, unsigned char* query, int queryLength, int idx, int idy, int idz)
{

	long long Count = 0L;
	//password = 70 61 73 73 77 6f 72 64
	int MessageLength = 8;
	unsigned char * Message = new unsigned char[MessageLength];
	Message[0] = idx;
	Message[1] = idy;
	Message[2] = idz;
	Message[3] = 0x73;
	Message[4] = 0x77;
	Message[5] = 0x6f;
	Message[6] = 0x72;
	Message[7] = 0x64;
	unsigned int * _stateSHA256 = new unsigned int[8];
	_stateSHA256[(0)] = 1779033703u;
	_stateSHA256[(1)] = 3144134277u;
	_stateSHA256[(2)] = 1013904242u;
	_stateSHA256[(3)] = 2773480762u;
	_stateSHA256[(4)] = 1359893119u;
	_stateSHA256[(5)] = 2600822924u;
	_stateSHA256[(6)] = 528734635u;
	_stateSHA256[(7)] = 1541459225u;

	int _KLen0 = 64;
	unsigned int * _K = new unsigned int[64];
	_K[(0)] = 1116352408u;
	_K[(1)] = 1899447441u;
	_K[(2)] = 3049323471u;
	_K[(3)] = 3921009573u;
	_K[(4)] = 961987163u;
	_K[(5)] = 1508970993u;
	_K[(6)] = 2453635748u;
	_K[(7)] = 2870763221u;
	_K[(8)] = 3624381080u;
	_K[(9)] = 310598401u;
	_K[(10)] = 607225278u;
	_K[(11)] = 1426881987u;
	_K[(12)] = 1925078388u;
	_K[(13)] = 2162078206u;
	_K[(14)] = 2614888103u;
	_K[(15)] = 3248222580u;
	_K[(16)] = 3835390401u;
	_K[(17)] = 4022224774u;
	_K[(18)] = 264347078u;
	_K[(19)] = 604807628u;
	_K[(20)] = 770255983u;
	_K[(21)] = 1249150122u;
	_K[(22)] = 1555081692u;
	_K[(23)] = 1996064986u;
	_K[(24)] = 2554220882u;
	_K[(25)] = 2821834349u;
	_K[(26)] = 2952996808u;
	_K[(27)] = 3210313671u;
	_K[(28)] = 3336571891u;
	_K[(29)] = 3584528711u;
	_K[(30)] = 113926993u;
	_K[(31)] = 338241895u;
	_K[(32)] = 666307205u;
	_K[(33)] = 773529912u;
	_K[(34)] = 1294757372u;
	_K[(35)] = 1396182291u;
	_K[(36)] = 1695183700u;
	_K[(37)] = 1986661051u;
	_K[(38)] = 2177026350u;
	_K[(39)] = 2456956037u;
	_K[(40)] = 2730485921u;
	_K[(41)] = 2820302411u;
	_K[(42)] = 3259730800u;
	_K[(43)] = 3345764771u;
	_K[(44)] = 3516065817u;
	_K[(45)] = 3600352804u;
	_K[(46)] = 4094571909u;
	_K[(47)] = 275423344u;
	_K[(48)] = 430227734u;
	_K[(49)] = 506948616u;
	_K[(50)] = 659060556u;
	_K[(51)] = 883997877u;
	_K[(52)] = 958139571u;
	_K[(53)] = 1322822218u;
	_K[(54)] = 1537002063u;
	_K[(55)] = 1747873779u;
	_K[(56)] = 1955562222u;
	_K[(57)] = 2024104815u;
	_K[(58)] = 2227730452u;
	_K[(59)] = 2361852424u;
	_K[(60)] = 2428436474u;
	_K[(61)] = 2756734187u;
	_K[(62)] = 3204031479u;
	_K[(63)] = 3329325298u;

	int _bufferLen0 = 64;
	unsigned char* _buffer = new unsigned char[64];
	for (int i = 0; i < 64; i++) _buffer[i] = 0;

	int _WLen0 = 64;
	unsigned int * _W = new unsigned int[64];
	for (int i = 0; i < 64; i++) _W[i] = 0;

	int partInLen = MessageLength;
	int partInBase = 0;
	int bufferLen = (int)(Count & 0x3f);

	Count += (long long)partInLen;

	if (bufferLen > 0 && bufferLen + partInLen >= 64)
	{
		CopyArray(Message, MessageLength, partInBase, _buffer, _bufferLen0, bufferLen, 64 - bufferLen);
		partInBase += 64 - bufferLen;
		partInLen -= 64 - bufferLen;
		SHATransform(_W, _WLen0, _stateSHA256, _buffer, _bufferLen0, _K, _KLen0);
		bufferLen = 0;
	}
	while (partInLen >= 64)
	{
		CopyArray(Message, MessageLength, partInBase, _buffer, _bufferLen0, 0, 64);
		partInBase += 64;
		partInLen -= 64;
		SHATransform(_W, _WLen0, _stateSHA256, _buffer, _bufferLen0, _K, _KLen0);
	}
	if (partInLen > 0)
	{
		CopyArray(Message, MessageLength, partInBase, _buffer, _bufferLen0, bufferLen, partInLen);
	}

	int padLen = 64 - (int)(Count & 0x3f);
	if (padLen <= 8)
	{
		padLen += 64;
	}

	unsigned char* pad = new unsigned char[padLen];
	pad[(0)] = 0x80;
	for (int i = 1; i < padLen; i++)
	{
		pad[i] = (unsigned char)0x00;
	}

	long long bitCount = Count * 8L;

	pad[padLen - 8] = (unsigned char)((int)bitCount >> 56 & 0xff);
	pad[padLen - 7] = (unsigned char)((int)bitCount >> 48 & 0xff);
	pad[padLen - 6] = (unsigned char)((int)bitCount >> 40 & 0xff);
	pad[padLen - 5] = (unsigned char)((int)bitCount >> 32 & 0xff);
	pad[padLen - 4] = (unsigned char)((int)bitCount >> 24 & 0xff);
	pad[padLen - 3] = (unsigned char)((int)bitCount >> 16 & 0xff);
	pad[padLen - 2] = (unsigned char)((int)bitCount >> 8 & 0xff);
	pad[padLen - 1] = (unsigned char)(bitCount & 0xff);

	bufferLen = 0;
	partInLen = padLen;
	partInBase = 0;
	bufferLen = (int)(Count & 0x3f);
	Count += (long long)partInLen;

	if (bufferLen > 0 && bufferLen + partInLen >= 64)
	{
		CopyArray(pad, padLen, partInBase, _buffer, _bufferLen0, bufferLen, 64 - bufferLen);
		partInBase += 64 - bufferLen;
		partInLen -= 64 - bufferLen;
		SHATransform(_W, _WLen0, _stateSHA256, _buffer, _bufferLen0, _K, _KLen0);
		bufferLen = 0;
	}
	while (partInLen >= 64)
	{
		CopyArray(pad, padLen, partInBase, _buffer, _bufferLen0, 0, 64);
		partInBase += 64;
		partInLen -= 64;
		SHATransform(_W, _WLen0, _stateSHA256, _buffer, _bufferLen0, _K, _KLen0);
	}
	if (partInLen > 0)
	{
		CopyArray(pad, padLen, partInBase, _buffer, _bufferLen0, bufferLen, partInLen);
	}

	unsigned char * ResultHash = new unsigned char[32];
	DWORDToBigEndian(ResultHash, _stateSHA256, 8);

	bool cont = true;
	int t = 0;
	while (cont)
	{
		if (ResultHash[t] == query[t])
		{
			cont = true;
			if (t == 31)
			{
				//full match found
				printf("%d : *** match found ***n", idx);
				for (int i = 0; i< 32; i++)
				{
					printf("%02X", ResultHash[i]);
					if (i < MessageLength)
					{
						output[i] = Message[i];
					}
				}
				printf("n");
				break;
			}
		}
		else
		{
			cont = false;
			break;
		}
		t++;
	}

	free(ResultHash);
	free(query);
	free(Message);
	free(_K);
	free(_W);
	free(_stateSHA256);
	free(_buffer);

	return output;
}

__device__ bool SHATransform(unsigned int* x, int xLen, unsigned int* state, unsigned char* block, int blockLen0, unsigned int* _K, int _KLen0)
{
	unsigned int num = state[(0)];
	unsigned int num2 = state[(1)];
	unsigned int num3 = state[(2)];
	unsigned int num4 = state[(3)];
	unsigned int num5 = state[(4)];
	unsigned int num6 = state[(5)];
	unsigned int num7 = state[(6)];
	unsigned int num8 = state[(7)];
	DWORDFromBigEndian(x, xLen, 16, block, blockLen0);
	for (int i = 16; i < 64; i++)
	{
		x[(i)] = sigma_1(x[(i - 2)]) + x[(i - 7)] + sigma_0(x[(i - 15)]) + x[(i - 16)];
	}
	for (int i = 0; i < 64; i++)
	{
		unsigned int num9 = num8 + Sigma_1(num5) + Ch(num5, num6, num7) + _K[(i)] + x[(i)];
		unsigned int num10 = num4 + num9;
		unsigned int num11 = num9 + Sigma_0(num) + Maj(num, num2, num3);
		i++;
		num9 = num7 + Sigma_1(num10) + Ch(num10, num5, num6) + _K[(i)] + x[(i)];
		unsigned int num12 = num3 + num9;
		unsigned int num13 = num9 + Sigma_0(num11) + Maj(num11, num, num2);
		i++;
		num9 = num6 + Sigma_1(num12) + Ch(num12, num10, num5) + _K[(i)] + x[(i)];
		unsigned int num14 = num2 + num9;
		unsigned int num15 = num9 + Sigma_0(num13) + Maj(num13, num11, num);
		i++;
		num9 = num5 + Sigma_1(num14) + Ch(num14, num12, num10) + _K[(i)] + x[(i)];
		unsigned int num16 = num + num9;
		unsigned int num17 = num9 + Sigma_0(num15) + Maj(num15, num13, num11);
		i++;
		num9 = num10 + Sigma_1(num16) + Ch(num16, num14, num12) + _K[(i)] + x[(i)];
		num8 = num11 + num9;
		num4 = num9 + Sigma_0(num17) + Maj(num17, num15, num13);
		i++;
		num9 = num12 + Sigma_1(num8) + Ch(num8, num16, num14) + _K[(i)] + x[(i)];
		num7 = num13 + num9;
		num3 = num9 + Sigma_0(num4) + Maj(num4, num17, num15);
		i++;
		num9 = num14 + Sigma_1(num7) + Ch(num7, num8, num16) + _K[(i)] + x[(i)];
		num6 = num15 + num9;
		num2 = num9 + Sigma_0(num3) + Maj(num3, num4, num17);
		i++;
		num9 = num16 + Sigma_1(num6) + Ch(num6, num7, num8) + _K[(i)] + x[(i)];
		num5 = num17 + num9;
		num = num9 + Sigma_0(num2) + Maj(num2, num3, num4);
	}
	state[(0)] += num;
	state[(1)] += num2;
	state[(2)] += num3;
	state[(3)] += num4;
	state[(4)] += num5;
	state[(5)] += num6;
	state[(6)] += num7;
	state[(7)] += num8;
	return true;
}

__device__ unsigned int RotateRight(unsigned int x, int n)
{
	return x >> (n & 31) | (int)x << (32 - n & 31);
}

__device__ unsigned int Ch(unsigned int x, unsigned int y, unsigned int z)
{
	return (x & y) ^ ((x ^ 4294967295u) & z);
}

__device__ unsigned int Maj(unsigned int x, unsigned int y, unsigned int z)
{
	return (x & y) ^ (x & z) ^ (y & z);
}

__device__ unsigned int sigma_0(unsigned int x)
{
	return RotateRight(x, 7) ^ RotateRight(x, 18) ^ x >> 3;
}

__device__ unsigned int sigma_1(unsigned int x)
{
	return RotateRight(x, 17) ^ RotateRight(x, 19) ^ x >> 10;
}

__device__ unsigned int Sigma_0(unsigned int x)
{
	return RotateRight(x, 2) ^ RotateRight(x, 13) ^ RotateRight(x, 22);
}

__device__ unsigned int Sigma_1(unsigned int x)
{
	return RotateRight(x, 6) ^ RotateRight(x, 11) ^ RotateRight(x, 25);
}



__device__ void DWORDToBigEndian(unsigned char* block, unsigned int* x, int digits)
{
	int i = 0;
	int num = 0;
	while (i < digits)
	{
		block[(num)] = (unsigned char)(x[(i)] >> 24 & 0xff);
		block[(num + 1)] = (unsigned char)(x[(i)] >> 16 & 0xff);
		block[(num + 2)] = (unsigned char)(x[(i)] >> 8 & 0xff);
		block[(num + 3)] = (unsigned char)(x[(i)] & 0xff);
		i++;
		num += 4;
	}
}

__device__ void DWORDFromBigEndian(unsigned int* x, int xLen0, int digits, unsigned char* block, int blockLen0)
{
	int i = 0;
	int num = 0;
	while (i < digits)
	{
		x[(i)] = (unsigned int)((int)block[(num)] << 24 | (int)block[(num + 1)] << 16 | (int)block[(num + 2)] << 8 | block[(num + 3)]);
		i++;
		num += 4;
	}
}

__device__ void CopyArray(unsigned char* SourceArray, int SourceArrayLen0, int SourceIndex, unsigned char* DestinationArray, int DestinationArrayLen0, int DestinationIndex, int Length)
{
	for (int i = 0; i < Length; i++)
	{
		DestinationArray[(i + DestinationIndex)] = SourceArray[(i + SourceIndex)];
	}
}
int main(void)
{
	printf("Copyright (C) 2014 Sean Bradley\n\n");
	printf("Permission is hereby granted, free of charge, to any person obtaining a copy of this software and\nassociated documentation files (the 'Software'),\nto deal in the Software without restriction,\nincluding without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense,\nand/or sell copies of the Software,\nand to permit persons to whom the Software is furnished to do so, subject to the following conditions:\n");
	printf("The above copyright notice and this permission notice shall be included in all copies or substantialnportions of the Software.\n");
	printf("n\THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT\nLIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.\nIN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,\nWHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE\nSOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.\n");
	printf("n\SeanWasEre\n\n");

	int N = 65535;
	int block_size = 16;
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);

	printf("%d %d\n", block_size, n_blocks);

	unsigned char *host_response, *dev_response;
	size_t size = 32 * sizeof(unsigned char);
	host_response = (unsigned char *)malloc(size);
	for (int i = 0; i < (int)size; i++)
	{
		host_response[i] = (unsigned char)0;
	}
	hipMalloc((void **)&dev_response, size);
	hipMemcpy(dev_response, host_response, size, hipMemcpyHostToDevice);

	printf("response buffer size=%d\n", size);
	//'password' as sha356 hash = 5e 88 48 98 da 28 04 71 51 d0 e5 6f 8d c6 29 27 73 60 3d 0d 6a ab bd d6 2a 11 ef 72 1d 15 42 d8
	unsigned char *host_query, *dev_query;
	size = 32 * sizeof(unsigned char);
	host_query = (unsigned char *)malloc(size);
	host_query[0] = 0x5e;
	host_query[1] = 0x88;
	host_query[2] = 0x48;
	host_query[3] = 0x98;
	host_query[4] = 0xda;
	host_query[5] = 0x28;
	host_query[6] = 0x04;
	host_query[7] = 0x71;
	host_query[8] = 0x51;
	host_query[9] = 0xd0;
	host_query[10] = 0xe5;
	host_query[11] = 0x6f;
	host_query[12] = 0x8d;
	host_query[13] = 0xc6;
	host_query[14] = 0x29;
	host_query[15] = 0x27;
	host_query[16] = 0x73;
	host_query[17] = 0x60;
	host_query[18] = 0x3d;
	host_query[19] = 0x0d;
	host_query[20] = 0x6a;
	host_query[21] = 0xab;
	host_query[22] = 0xbd;
	host_query[23] = 0xd6;
	host_query[24] = 0x2a;
	host_query[25] = 0x11;
	host_query[26] = 0xef;
	host_query[27] = 0x72;
	host_query[28] = 0x1d;
	host_query[29] = 0x15;
	host_query[30] = 0x42;
	host_query[31] = 0xd8;
	hipMalloc((void **)&dev_query, size);
	hipMemcpy(dev_query, host_query, size, hipMemcpyHostToDevice);

	std::clock_t start;

	start = std::clock();
	//kernel(host_response, 32, dev_query, 32, N);
	kernel <<< n_blocks, block_size >>> (dev_response, dev_query, 32, N);
	hipMemcpy(host_response, dev_response, sizeof(unsigned char) * 32, hipMemcpyDeviceToHost);
	printf("ms = %lu.nn", (std::clock() - start));

	hipFree(dev_response); hipFree(dev_query);

	for (int j = 0; j < 32; j++)
	{
		//printf("%02X", host_response[j + (i * 20)]);
		printf("%02X", host_response[j]);
	}
	printf("n");

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%sn", hipGetErrorString(err));

	free(host_response);

	printf("\nCopyright SeanWasEre.com 2014\n\n");

	system("pause");

}
